#include "hip/hip_runtime.h"
// @file im2row_gpu.cu
// @brief Stack image patches as matrix rows (GPU)
// @author Andrea Vedaldi

/*
Copyright (C) 2014-15 Andrea Vedaldi.
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

#include "im2row.hpp"
#include "../datacu.hpp"
#include <iostream>
#include <string>

using namespace vl ;

/* ---------------------------------------------------------------- */
/*                                                           im2row */
/* ---------------------------------------------------------------- */

template <typename T> __global__ void
im2row_gpu_kernel(T* stacked,
                  T const* data,
                  const int numPatchesX,
                  const int numPatchesY,
                  const int numPatchSlices,
                  const int width,
                  const int height,
                  const int windowWidth,
                  const int windowHeight,
                  const int strideX,
                  const int strideY,
                  const int padLeft,
                  const int padTop,
                  const int holeX,
                  const int holeY)
{
  /* each kernel copies the pixels in an image patch for one channel */
  int index = threadIdx.x + blockIdx.x * blockDim.x ;
  if (index < numPatchSlices) {
    /*
     get the patch slice (x,y,z) to copy
     */
    int x = index ;
    int y = x / numPatchesX ;
    int z = y / numPatchesY ;   // c_im
    x %= numPatchesX ;  // w in gpapan
    y %= numPatchesY ;  // h in gpapan

    /*
     pick the top-left corner of the patch slice in the input image
     */
    int x_data = x * strideX - padLeft ;    // w_im
    int y_data = y * strideY - padTop ;     // h_im
    data += (z * height + y_data) * width + x_data ;

    /*
     pick the column of the stacked image which contains this patch,
     and move down along the column at the beginning of the patch slice
     */
    int patchSliceOffset = (windowWidth*windowHeight) * z ; // c
    stacked += (numPatchesY * patchSliceOffset + y) * numPatchesX + x ; // data_col

    /*
     copy the patch slice
     */
    for (int v = 0 ; v < windowHeight ; ++v) {
      for (int u = 0 ; u < windowWidth ; ++u) {
        if (y_data + v*holeY >= 0 &&
            y_data + v*holeY < height &&
            x_data + u*holeX >= 0 &&
            x_data + u*holeX < width) {
          *stacked = data[v * holeY * width + u * holeX] ;
        } else {
          *stacked = 0 ;
        }
        stacked += (numPatchesX*numPatchesY) ;
      }
    }
  }
}

template <typename T> static inline hipError_t
im2row_gpu(T* stacked,
           T const* data,
           size_t width,
           size_t height,
           size_t depth,
           size_t windowWidth,
           size_t windowHeight,
           size_t strideX,
           size_t strideY,
           size_t padLeft,
           size_t padRight,
           size_t padTop,
           size_t padBottom,
           size_t holeX,
           size_t holeY)
{
  /* Each kernel instance copies a feature dimension of a patch */
  int windowHeightEff = windowHeight + (windowHeight-1) * (holeY - 1);  //(kernel_h_eff in gpapan code)
  int windowWidthEff  = windowWidth  + (windowWidth-1)  * (holeX - 1);  //(kernel_w_eff in gpapan code)
  int numPatchesX = (width + (padLeft + padRight) - windowWidthEff)/strideX + 1 ;
  int numPatchesY = (height + (padTop + padBottom) - windowHeightEff)/strideY + 1 ;
  int numPatchSlices = numPatchesX * numPatchesY * depth ;

  im2row_gpu_kernel<T>
  <<< divideUpwards(numPatchSlices, VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
  (stacked,
   data,
   numPatchesX,
   numPatchesY,
   numPatchSlices,
   width, height,
   windowWidth, windowHeight,
   strideX, strideY,
   padLeft, padTop,
   holeX, holeY) ;

  return hipPeekAtLastError() ;
}


template <> vl::Error
vl::impl::im2row<vl::GPU, float>(vl::Context& context,
                                 float* stacked,
                                 float const* data,
                                 size_t height, size_t width, size_t depth,
                                 size_t windowHeight, size_t windowWidth,
                                 size_t strideY, size_t strideX,
                                 size_t padTop, size_t padBottom,
                                 size_t padLeft, size_t padRight,
                                 size_t holeX, size_t holeY)
{
  int status ;
  status = im2row_gpu<float>(stacked, data,
                             height, width, depth,
                             windowHeight, windowWidth,
                             strideY, strideX,
                             padTop, padBottom, padLeft, padRight,holeX,holeY) ;
  return (status == hipSuccess) ? vl::vlSuccess : vl::vlErrorCuda ;
}


/* ---------------------------------------------------------------- */
/*                                                           row2im */
/* ---------------------------------------------------------------- */

template <typename T>
__global__ void row2im_gpu_kernel(T* data,
                                  T const* stacked,
                                  const int numPatchesX,
                                  const int numPatchesY,
                                  const int dataVolume,
                                  const int width,
                                  const int height,
                                  const int depth,
                                  const int windowWidth,
                                  const int windowHeight,
                                  const int strideX,
                                  const int strideY,
                                  const int padLeft,
                                  const int padTop,
                                  const int holeX,
                                  const int holeY)
{
  // row2im works in the inverse way than the corresponding caffe function:
  // for each point in the output image, we find all points in patches that contribute
  // and accumulate them.
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < dataVolume)
  {
    T accumulator = 0 ;
    /*
     This kernel accumulates on data[index] all elements in stacked
     that receive copies of data[index] in im2row.

     Consider coordinate (x_data,y_data) in the input image. Relative to patch
     (x,y), this has offset

     u = x_data - (x * strideX - padLeft)
     v = y_data - (y * strideY - padRight)

     In particular, (x_data,y_data) is contained (and hence contributes)
     to patch (x,y) if, and only if,

     0 <= u < windowWidth  <==>  1) x_data >= x * strideX - padLeft
     2) x_data <  x * strideX - padLeft + windowWidth

     and similar for y.

     Hence, the patches that contribute to (x_data,y_d ata) are given
     by indexes (x,y) such that

     (x_data + padLeft - windowWidth)/stride < x
     <= (x_data + padLeft)/stride

     or, accounting for the boundaries,

     x1 <= x <= x2, such that
     x1 = max(0,  1 + floor(x_data + padLeft - windowWidth)/stride),
     x2 = min(numPatchesX-1,  floor(x_data + padLeft)/stride),

     and similar for y.

     Note that (x_data + padLeft - windowWidth) may be negative. In this case,
     the C convention for rounding division towards zero fails to compute
     the floor() properly. Instead, we check this case explicitly and set
     */

    int x_data = index ;
    int y_data = x_data / width ;
    int z = y_data / height ;
    x_data %= width ;
    y_data %= height ;

    // x1,x2,y1,y2 is the range of the
    // patches that contribute to data[index]. For each one of those patches we
    // also have to find exactly which pixels in the patch contribute, taking
    // holes into account.
    int windowHeightEff = windowHeight + (windowHeight-1) * (holeY - 1);  //(kernel_h_eff in gpapan code)
    int windowWidthEff  = windowWidth  + (windowWidth-1)  * (holeX - 1);  //(kernel_w_eff in gpapan code)
    int dx = x_data + padLeft - windowWidthEff ;
    int dy = y_data + padTop  - windowHeightEff ;
    int x1 = (dx >= 0) ? dx/strideX + 1 : 0 ;
    int y1 = (dy >= 0) ? dy/strideY + 1 : 0 ;
    int x2 = min((x_data + padLeft) / strideX, numPatchesX - 1) ;
    int y2 = min((y_data + padTop) / strideY, numPatchesY - 1) ;


    /*
     Knowing which patches (x,y) contribute to (x_data,y_data) is not enough;
     we need to determine the specific element within each patch. This
     is given by the offset as given above:

     u(x) = x_data - (x * strideX - padLeft)
     v(y) = y_data - (y * strideY - padRight)

     Now we can compute the indeces of the elements of stacked[] to accumulate:

     stackedIndex(x,y) =
     (y * numPatchesX + x) +                 // column offset
     ((z * windowHeight + v(y)) * windowWidth + u(x)) *  // within patch offset
     (numPatchesX*numPatchesY)

     Substituting the expression fo u(x), we find

     stackedIndex(x,y) =
     = (y * numPatchesX + x)
     + ((z * windowHeight + y_data + padTop) * windowWidth + x_data + padLeft)
     * (numPatchesX*numPatchesY)
     - ((y * strideY) * windowWidth + x * strideX)
     * (numPatchesX*numPatchesY)
     = (z * windowHeight + y_data + padTop) * windowWidth + x_data + padLeft)
     + x * (1 - strideX*numPatchesY*numPatchesX)
     + y * (1 - strideY*numPatchesY*windowWidth)*numPatchesX ;

     */

    // stacked is effectively a nPatches x nPixelsPerPatch array
//    int deltax = (1 - strideX * numPatchesY * numPatchesX) ;
//    int deltay = (1 - strideY * numPatchesY * windowWidth) * numPatchesX ;
//    stacked += ((z * windowHeight + y_data + padTop) * windowWidth + (x_data + padLeft)) * (numPatchesX*numPatchesY) ;
//    for (int y = y1 ; y <= y2 ; ++y) {
//      for (int x = x1 ; x <= x2 ; ++x) {
//          accumulator += stacked[y * deltay + x * deltax];
//      }
//    }

    for (int y = y1 ; y <= y2 ; ++y) {
      for (int x = x1 ; x <= x2 ; ++x) {
        int vy = y_data - y*strideY + padTop;
        int ux = x_data - x*strideX + padLeft;
        bool isHole = (ux > 0 && ux <= holeX) || (vy > 0 && vy <= holeY) ||
                      ((ux % (holeX+1)) > 0)  || ((vy % (holeY+1)) > 0);
        if (!isHole) {
          int stackIndex = (y * numPatchesX + x) +                // column offset
                  ((z * windowHeight + vy) * windowWidth + ux) *  // within patch offset
                  (numPatchesX*numPatchesY);
          accumulator += stacked[stackIndex];
        }
      }
    }
    data[index] = accumulator;
  }
}

template <typename T> static inline hipError_t
row2im_gpu(T* data,
           T const* stacked,
           size_t width,
           size_t height,
           size_t depth,
           size_t windowWidth,
           size_t windowHeight,
           size_t strideX,
           size_t strideY,
           size_t padLeft,
           size_t padRight,
           size_t padTop,
           size_t padBottom,
           size_t holeX,
           size_t holeY)
{
  /*
   Each kernel integrates all contributions to a particular element
   of data.
   */

  int windowHeightEff = windowHeight + (windowHeight-1) * (holeY - 1);  //(kernel_h_eff in gpapan code)
  int windowWidthEff  = windowWidth  + (windowWidth-1)  * (holeX - 1);  //(kernel_w_eff in gpapan code)
  int numPatchesX = (width + (padLeft + padRight) - windowWidthEff)/strideX + 1 ;
  int numPatchesY = (height + (padTop + padBottom) - windowHeightEff)/strideY + 1 ;
  int dataVolume = width * height * depth ;

//  std::cout << "width: " << width << " height: " << height << std::endl;
//  std::cout << "windowWidth: " << windowWidth << " windowHeight: " << windowHeight << std::endl;
//  std::cout << "windowWidthEff: " << windowWidthEff << " windowHeightEff: " << windowHeightEff << std::endl;


  row2im_gpu_kernel<T>
  <<< divideUpwards(dataVolume, VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
  (data,
   stacked,
   numPatchesX,
   numPatchesY,
   dataVolume,
   width, height, depth,
   windowWidth, windowHeight,
   strideX, strideY,
   padLeft, padTop,
   holeX, holeY);

  return hipPeekAtLastError() ;
}

template <> vl::Error
vl::impl::row2im<vl::GPU, float>(vl::Context& context,
                                 float* data,
                                 float const* stacked,
                                 size_t height, size_t width, size_t depth,
                                 size_t windowHeight, size_t windowWidth,
                                 size_t strideY, size_t strideX,
                                 size_t padTop, size_t padBottom,
                                 size_t padLeft, size_t padRight,
                                 size_t holeX, size_t holeY)
{
  int status ;
  status = row2im_gpu<float>(data, stacked,
                             height, width, depth,
                             windowHeight, windowWidth,
                             strideY, strideX,
                             padTop, padBottom, padLeft, padRight,holeX,holeY) ;
  return (status == hipSuccess) ? vl::vlSuccess : vl::vlErrorCuda ;
}
